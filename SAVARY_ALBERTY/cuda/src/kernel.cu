#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "particule.h"

__global__ void kernel_acceleration( int n, particule_t * in) {
	float sumX, sumY, sumZ ,dX, dY, dZ, distance, masse_invDist3;
	int i;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if ( index < n ) { 
		sumX = 0;
		sumY = 0;
		sumZ = 0;
		for (i = 0 ; i < NB_PARTICULE ; i++){
			if (i != index){
				dX = in[i].PosX - in[index].PosX;
				dY = in[i].PosY - in[index].PosY;
				dZ = in[i].PosZ - in[index].PosZ;

				distance = sqrtf( Pow2(dX) + Pow2(dY) + Pow2(dZ) );
				if ( distance < 1.0 ) distance = 1.0;

				masse_invDist3 = in[i].Masse * (1/Pow3(distance)) * ME;

				sumX += dX * masse_invDist3;
				sumY += dY * masse_invDist3;
				sumZ += dZ * masse_invDist3;
			}
		}

		in[index].VelX += sumX;
		in[index].VelY += sumY;
		in[index].VelZ += sumZ;
	}
}

__global__ void kernel_actualisation(int n, particule_t * in) {
	float g_t = 0.1f;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	in[index].PosX += in[index].VelX * g_t;
	in[index].PosY += in[index].VelY * g_t;
	in[index].PosZ += in[index].VelZ * g_t;
}

void cuda_calcul_acceleration( int nblocks, int nthreads, int n, particule_t * in ) {
	kernel_acceleration<<<nblocks, nthreads>>>( n, in);
	kernel_actualisation<<<nblocks, nthreads>>>( n, in);
}



